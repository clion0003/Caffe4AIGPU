#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
  __global__ void mat_mul_N_N(int m, int k ,int n, float alpha, float beta, const float *a, const float *b, float *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[row*k + index] * b[index*n + col];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void mat_mul_N_T(int m, int k ,int n, float alpha, float beta, const float *a, const float *b, float *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[row*k + index] * b[col*k + index];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void mat_mul_T_N(int m, int k ,int n, float alpha, float beta, const float *a, const float *b, float *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[index*m + row] * b[index*n + col];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void mat_mul_T_T(int m, int k ,int n, float alpha, float beta, const float *a, const float *b, float *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[index*m + row] * b[col*k + index];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  
  __global__ void mat_vec_N(int m, int n, float alpha, float beta, const float *a, const float *x ,float *y) {
    // calculate the row & col index of the element
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m) {
      // do dot product between row of a and col of b
      for (int index = 0; index < n; index++)
        result += a[row*n + index] * x[index];
  
      y[row] = alpha * result + beta * y[row];
  
    }
  }
  
  __global__ void mat_vec_T(int m, int n, float alpha, float beta, const float *a, const float *x ,float *y) {
    // calculate the row & col index of the element
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    float result = 0;
    if (row < m) {
      // do dot product between row of a and col of b
      for (int index = 0; index < n; index++)
        result += a[index*m + row] * x[index];
  
      y[row] = alpha * result + beta * y[row];
  
    }
  }

  __global__ void double_mat_mul_N_N(int m, int k ,int n, double alpha, double beta, const double *a, const double *b, double *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[row*k + index] * b[index*n + col];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void double_mat_mul_N_T(int m, int k ,int n, double alpha, double beta, const double *a, const double *b, double *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[row*k + index] * b[col*k + index];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void double_mat_mul_T_N(int m, int k ,int n, double alpha, double beta, const double *a, const double *b, double *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[index*m + row] * b[index*n + col];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  __global__ void double_mat_mul_T_T(int m, int k ,int n, double alpha, double beta, const double *a, const double *b, double *ab) {
    // calculate the row & col index of the element
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m && col < n) {
      // do dot product between row of a and col of b
      for (int index = 0; index < k; index++)
        result += a[index*m + row] * b[col*k + index];
  
      ab[row*n + col] = alpha * result + beta * ab[row*n + col];
  
    }
  }
  
  
  __global__ void double_mat_vec_N(int m, int n, double alpha, double beta, const double *a, const double *x ,double *y) {
    // calculate the row & col index of the element
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m) {
      // do dot product between row of a and col of b
      for (int index = 0; index < n; index++)
        result += a[row*n + index] * x[index];
  
      y[row] = alpha * result + beta * y[row];
  
    }
  }
  
  __global__ void double_mat_vec_T(int m, int n, double alpha, double beta, const double *a, const double *x ,double *y) {
    // calculate the row & col index of the element
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    double result = 0;
    if (row < m) {
      // do dot product between row of a and col of b
      for (int index = 0; index < n; index++)
        result += a[index*m + row] * x[index];
  
      y[row] = alpha * result + beta * y[row];
  
    }
  }

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  // int lda = (TransA == CblasNoTrans) ? K : M;
  // int ldb = (TransB == CblasNoTrans) ? N : K;
  //hipblasOperation_t cuTransA =
  //    (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  //hipblasOperation_t cuTransB =
  //    (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
      
	dim3 block(32, 32);
  dim3 grid((N + 31) / 32, (M + 31) / 32);
  if(TransA == CblasNoTrans){
    if(TransB == CblasNoTrans) mat_mul_N_N << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
    else mat_mul_N_T << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
  }
  else{
    if(TransB == CblasNoTrans) mat_mul_T_N << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
    else mat_mul_T_T << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
  }
  //CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
  //    N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
	dim3 block(32, 32);
  dim3 grid((N + 31) / 32, (M + 31) / 32);
  if(TransA == CblasNoTrans){
    if(TransB == CblasNoTrans) double_mat_mul_N_N << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
    else double_mat_mul_N_T << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
  }
  else{
    if(TransB == CblasNoTrans) double_mat_mul_T_N << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
    else double_mat_mul_T_T << <grid, block >> > (M, K, N, alpha, beta, A, B, C);
  }
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  //hipblasOperation_t cuTransA =
  //    (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  //CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
  //    A, N, x, 1, &beta, y, 1));
	dim3 block(256);
  dim3 grid((N + 255) / 256);
  if(TransA == CblasNoTrans) mat_vec_N <<< grid, block >>> (M, N, alpha, beta, A, x, y);
  else mat_vec_T <<< grid, block >>> (M, N, alpha, beta, A, x, y);
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
//  hipblasOperation_t cuTransA =
//      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
//  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
//      A, N, x, 1, &beta, y, 1));
dim3 block(256);
dim3 grid((N + 255) / 256);
if(TransA == CblasNoTrans) double_mat_vec_N <<< grid, block >>> (M, N, alpha, beta, A, x, y);
else double_mat_vec_T <<< grid, block >>> (M, N, alpha, beta, A, x, y);
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  //CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  //CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  //CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  //CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  //hipStream_t initial_stream;
  //CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  //CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  //CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  //CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  //hipStream_t initial_stream;
  //CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  //CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  //CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  //CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  //CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  //CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  //CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  //CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  //CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  //CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  //CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  //CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  //CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  //CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  //const float range = b - a;
  //if (range != static_cast<float>(1)) {
  //  caffe_gpu_scal(n, range, r);
  //}
  //if (a != static_cast<float>(0)) {
  //  caffe_gpu_add_scalar(n, a, r);
  //}
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  //CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  //const double range = b - a;
  //if (range != static_cast<double>(1)) {
  //  caffe_gpu_scal(n, range, r);
  //}
  //if (a != static_cast<double>(0)) {
  //  caffe_gpu_add_scalar(n, a, r);
  //}
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  //CURAND_CHECK(
  //    hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  //CURAND_CHECK(
  //    hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
